
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK(e) { int res = (e); if (res) printf("CUDA ERROR %d\n", res); }

#define CHANNEL 3
#define BLOCK_DIM_X (30)
#define BLOCK_DIM_Y (30)
#define AUGMENTED_X (BLOCK_DIM_X+2)
#define AUGMENTED_Y (BLOCK_DIM_Y+2)

struct Image {
  int width;
  int height;
  unsigned int bytes;
  unsigned char *img;
  unsigned char *dev_img;
};


// Reads a color PPM image file (name provided), and
// saves data in the provided Image structure.
// The max_col_val is set to the value read from the
// input file. This is used later for writing output image.
int readInpImg (const char * fname, Image & source, int & max_col_val){

  FILE *src;

  if (!(src = fopen(fname, "rb")))
  {
      printf("Couldn't open file %s for reading.\n", fname);
      return 1;
  }

  char p,s;
  fscanf(src, "%c%c\n", &p, &s);
  if (p != 'P' || s != '6')   // Is it a valid format?
  {
      printf("Not a valid PPM file (%c %c)\n", p, s);
      exit(1);
  }

  fscanf(src, "%d %d\n", &source.width, &source.height);
  fscanf(src, "%d\n", &max_col_val);

  int pixels = source.width * source.height;
  source.bytes = pixels * 3;  // 3 => colored image with r, g, and b channels
  source.img = (unsigned char *)malloc(source.bytes);
  if (fread(source.img, sizeof(unsigned char), source.bytes, src) != source.bytes)
    {
       printf("Error reading file.\n");
       exit(1);
    }
  fclose(src);
  return 0;
}

// Write a color image into a file (name provided) using PPM file format.
// Image structure represents the image in the memory.
int writeOutImg(const char * fname, const Image & roted, const int max_col_val){

  FILE *out;
  if (!(out = fopen(fname, "wb")))
  {
      printf("Couldn't open file for output.\n");
      return 1;
  }
  fprintf(out, "P6\n%d %d\n%d\n", roted.width, roted.height, max_col_val);
  if (fwrite(roted.img, sizeof(unsigned char), roted.bytes , out) != roted.bytes)
  {
      printf("Error writing file.\n");
      return 1;
  }
  fclose(out);
  return 0;
}

__global__ void blur(unsigned char * in,unsigned char* out,int w,int h){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (col < w && row < h){
        int pixValR = 0;
        int pixValG = 0;
        int pixValB = 0;
        int count = 0;
        for(int dr = -1;dr <=1;dr++){
            for(int dc = -1;dc <=1;dc++){
                int curRow = row + dr;
                int curCol = col + dc;
                if (curRow >=0 && curRow < h && curCol >=0 && curCol < w){
                    int rgbOffset = (curRow*w + curCol)*3;
                    pixValR += in[rgbOffset];
                    pixValG += in[rgbOffset+1];
                    pixValB += in[rgbOffset+2];
                    count++;
                }
            }
        }
        int offset = (row*w+col)*3;
        out[offset] = (unsigned char)(pixValR/count);
        out[offset+1] = (unsigned char)(pixValG/count);
        out[offset+2] = (unsigned char)(pixValB/count);
    }
}
__global__ void blurShared(unsigned char* in, unsigned char* out, int w, int h) {
    int filterRow, filterCol;
    int cornerRow, cornerCol;
    int tx = threadIdx.x; int ty = threadIdx.y;
    int bx = blockIdx.x; int by = blockIdx.y;
    int bdx = blockDim.x; int bdy = blockDim.y;
    int row = by * (bdy - 2) + ty;
    int col = bx * (bdx - 2) + tx;
    if ((row < h + 1) && (col < w + 1)) {

        __shared__ unsigned char tile[AUGMENTED_Y][AUGMENTED_X][3];

        // load into shared memory
        int imgRow = row - 1;
        int imgCol = col - 1;
        if ((imgRow < h) && (imgCol < w) && (imgRow >= 0) && (imgCol >= 0)) {
            int rgbOffset = (imgRow*w + imgCol)*3;
            tile[ty][tx][0] = in[rgbOffset];
            tile[ty][tx][1] = in[rgbOffset+1];
            tile[ty][tx][2] = in[rgbOffset+2];
        }
        else {
            tile[ty][tx][0] = 0;
            tile[ty][tx][1] = 0;
            tile[ty][tx][2] = 0;
        }

        __syncthreads();

        int pixValR = 0;
        int pixValG = 0;
        int pixValB = 0;
        int count = 0;

        
        if ((tx >= 1) && (ty >= 1) && (ty < bdy - 1) && (tx < bdx - 1)) {

            cornerRow = ty - 1;
            cornerCol = tx - 1;

            for (int i = 0; i < 3; i++) {
                for (int j = 0; j < 3; j++) {
                    filterRow = cornerRow + i;
                    filterCol = cornerCol + j;

                    if ((filterRow >= 0) && (filterRow <= h) && (filterCol >= 0) && (filterCol <= w)) {
                        pixValR += tile[filterRow][filterCol][0];
                        pixValG += tile[filterRow][filterCol][1];
                        pixValB += tile[filterRow][filterCol][2];
                        count++;
                    }
                }
            }
            int offset = (imgRow*w + imgCol)*3;
            out[offset] = (unsigned char)(pixValR/count);
            out[offset+1] = (unsigned char)(pixValG/count);
            out[offset+2] = (unsigned char)(pixValB/count);
        }
    }
}
int main(int argc, char **argv)
{

  if (argc != 2)
  {
      printf("Usage: exec filename\n");
      exit(1);
  }
  char *fname = argv[1];
  Image source;
  Image source1;
  int max_col_val;
  if (readInpImg(fname, source, max_col_val) != 0)  exit(1);
  if (readInpImg(fname, source1, max_col_val) != 0)  exit(1);
  unsigned char *d_img;
  unsigned char *d_img_res;
  unsigned char *d_img_res1;
  int size = source.bytes * sizeof(char);
  hipMalloc((void **)&d_img,size);
  hipMalloc((void **)&d_img_res,size);
  hipMalloc((void **)&d_img_res1,size);
  hipMemcpy(d_img,source.img,size,hipMemcpyHostToDevice);
  dim3 dimGrid(ceil(1.0*source.width/BLOCK_DIM_X),ceil(1.0*source.height/BLOCK_DIM_Y),1);
  dim3 dimBlock(BLOCK_DIM_X,BLOCK_DIM_Y,1);
  hipEvent_t start,stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start,0);
  blur<<<dimGrid,dimBlock>>>(d_img,d_img_res,source.width,source.height);
  hipMemcpy(source.img,d_img_res,size,hipMemcpyDeviceToHost);
  hipEventRecord(stop,0);
  float costtime;
  hipEventSynchronize(stop);
  hipEventElapsedTime(&costtime,start,stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  printf("image size (%d,%d) with block dim:(%d,%d) common using time: %f\n",source.width,source.height,BLOCK_DIM_X,BLOCK_DIM_Y,costtime);
  dim3 dimGridShared(ceil(1.0*source.width/BLOCK_DIM_X),ceil(1.0*source.height/BLOCK_DIM_Y),1);
  dim3 dimBlockShared(AUGMENTED_X,AUGMENTED_Y,1);
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start,0);
  blurShared<<<dimGridShared,dimBlockShared>>>(d_img,d_img_res1,source.width,source.height);
  //for(int i=0;i<5;i++){//test the change of RGB value
  //for(int j=0;j<5;j++){
  //printf("%d ",source.img[(i*source.width+j)*3]);
  //}
  //printf("\n");
  //}
  //printf("\n");
  hipMemcpy(source1.img,d_img_res1,size,hipMemcpyDeviceToHost);
  hipEventRecord(stop,0);
  float costtime1;
  hipEventSynchronize(stop);
  hipEventElapsedTime(&costtime1,start,stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  printf("image size (%d,%d) with block dim:(%d,%d) shared using time: %f\n",source.width,source.height,BLOCK_DIM_X,BLOCK_DIM_Y,costtime1);
  //for(int i=0;i<5;i++){//test the change of RGB value
  //for(int j=0;j<5;j++){
  //printf("%d ",source.img[(i*source.width+j)*3]);
  //}
  //printf("\n");
  //}
  //printf("\n");
  for(int i=1;i<source.height-1;i++){
    for(int j=1;j<source.width-1;j++){
      if(source.img[(i*source.width+j)*3]!=source1.img[(i*source.width+j)*3]){
          printf("%d,%d,%d\n",i,source.img[i],source1.img[i]);
          break;
      }
    }
  }

  // Write the output file
  if (writeOutImg("blur.ppm", source, max_col_val) != 0) // For demonstration, the input file is written to a new file named "roted.ppm"
   exit(1);

  free(source.img);

  exit(0);
}
